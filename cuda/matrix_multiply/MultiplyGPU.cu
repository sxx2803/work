#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <iostream>
#include "common.h"

// GPU Kernel to perform a single inner product
__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int size)
{
	// Retrieve our coordinates in the block
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	// Temporary result
	float p = 0;
	// Perform inner product
	for (int k = 0; k < size; k++) {
		p += Md[ty * size + k] * Nd[k * size + tx];
	}
	// Write to result
	Pd[ty * size + tx] = p;
}

// C Function to run matrix multiplication kernel
bool MatrixMultiplicationGPU(float* M, float* N, float* P, int size)
{
	// Error return value
	hipError_t status;

	// Number of bytes in the matrix.
	int bytes = size * size * sizeof(float);

	// Pointers to the device arrays
	float *Md, *Nd, *Pd;

	// Allocate memory on the device to store each matrix
	hipMalloc((void**) &Md, bytes);
	hipMalloc((void**) &Nd, bytes);
	hipMalloc((void**) &Pd, bytes);

	// Copy the host input data to the device
	hipMemcpy(Md, M, bytes, hipMemcpyHostToDevice);
	hipMemcpy(Nd, N, bytes, hipMemcpyHostToDevice);

	// Specify the size of the grid and the size of the block
	dim3 dimBlock(size, size);	// Matrix is contained in a block
	dim3 dimGrid(1, 1);			// Only using a single grid element today

	// Launch the kernel on a size-by-size block of threads
	MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, size);

	// Wait for completion
	hipDeviceSynchronize();

	// Check for errors
	status = hipGetLastError();
	if (status != hipSuccess) {
		std::cout << "Kernel failed: " << hipGetErrorString(status) << std::endl;
		hipFree(Md);
		hipFree(Nd);
		hipFree(Pd);
		return false;
	}

	// Retrieve the result matrix
	hipMemcpy(P, Pd, bytes, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(Md);
	hipFree(Nd);
	hipFree(Pd);

	// Success
	return true;
}